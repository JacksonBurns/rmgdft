
#include <hip/hip_runtime.h>
/*
 *
 * Copyright 2018 The RMG Project Developers. See the COPYRIGHT file 
 * at the top-level directory of this distribution or in the current
 * directory.
 * 
 * This file is part of RMG. 
 * RMG is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 2 of the License, or
 * any later version.
 *
 * RMG is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
*/


#if CUDA_ENABLED
#include <cuda.h>
#include <cuda_runtime.h>
#include <cuda_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <crt/host_runtime.h>
#include <cublas_v2.h>
#include "ErrorFuncs.h"

__global__ void gramsch_update_psi_kernel(
                                     double *V,
                                     double *C,
                                     double *G,
                                     int n,
                                     int eig_start,
                                     int eig_stop)
{

    int ix = (threadIdx.x + eig_start) * n + blockIdx.x;
    int iy = blockIdx.x * n + threadIdx.x + eig_start;
    __syncthreads();
    G[ix] = V[iy];
    __syncthreads();
}

//for(int st1=eig_start;st1<eig_stop;st1++)
//{
//    for(int st2=0;st2<n;st2++)
//    {
//        G[st1*n + st2] = V[st1 + st2*n];
//    }
//}
//memcpy(&V[eig_start*n], &G[eig_start*n], eig_step*n*sizeof(KpointType));


void gramsch_update_psi(double *V,
                        double *C,
                        int N,
                        int eig_start,
                        int eig_stop,
                        cublasHandle_t cublasH)
{

    int eig_step = eig_stop - eig_start;
    int ione = 1;
    double alpha = -1.0;

    // We get the inverse of the diagonal elements here rather than inside the loop to avoid page faults
    double *darr;
    RmgCudaError(__FILE__, __LINE__, cudaMallocManaged ( &darr, N*sizeof(double), cudaMemAttachGlobal ), "Error: cudaMallocManaged failed.\n");
    for(int i = 0;i < N;i++) darr[i] = 1.0 / C[i*N + i];
    //cublasDcopy(cublasH, N, C, N + 1, darr, 1);
    //for(int i = 0;i < N;i++) darr[i] = 1.0 / darr[i];
    DeviceSynchronize();
    /* apply inverse of cholesky factor to states */
    for (int st = 0; st < N; st++)
    {

        /* normalize V[st] */
        cublasDscal(cublasH, eig_step, &darr[st], &V[st * N + eig_start], ione);

        /* subtract the projection along c[st] from the remaining vectors */
        int idx = N - st - 1;
        if(idx)
        {
            cublasDger(cublasH, eig_step, idx, &alpha, &V[st * N + eig_start], ione,
               &C[(st+1) + N*st], ione, &V[(st+1) * N + eig_start], N);
        }

    } /* end of for */

    DeviceSynchronize();
    cudaFree(darr);
}

#endif

